#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

typedef struct {
    int degree;
    double *coefficients;
} Polynomial;


__global__ void initialize_curand(hiprandState *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), idx, 0, &state[idx]);
}

__global__ void generate_random_polynomial_kernel(int max_degree, double range, Polynomial *poly, hiprandState *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx <= poly->degree) {
        double rand_val = hiprand_uniform_double(&state[idx]);
        poly->coefficients[idx] = (rand_val * 2 * range - range);
    }
}

Polynomial *generate_random_polynomial(int max_degree, double range) {
    int degree = rand() % (max_degree - 1) + 1;

    Polynomial *poly = (Polynomial *)malloc(sizeof(Polynomial));
    poly->degree = degree;

    hipMalloc((void **)&poly->coefficients, (degree + 1) * sizeof(double));

    hiprandState *state;
    hipMalloc((void **)&state, (degree + 1) * sizeof(hiprandState));

    int blockSize = 256;
    int gridSize = (degree + blockSize - 1) / blockSize;

    initialize_curand<<<gridSize, blockSize>>>(state);
    generate_random_polynomial_kernel<<<gridSize, blockSize>>>(max_degree, range, poly, state);

    hipFree(state);

    return poly;
}



__global__ void print_polynomial_kernel(Polynomial *poly, double *coefficients_host) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx <= poly->degree) {
        coefficients_host[idx] = poly->coefficients[idx];
    }
}

void print_polynomial(Polynomial *poly) {
    double *coefficients_host = (double *)malloc((poly->degree + 1) * sizeof(double));
    
    int blockSize = 256;
    int gridSize = (poly->degree + blockSize - 1) / blockSize;
    
    print_polynomial_kernel<<<gridSize, blockSize>>>(poly, coefficients_host);
    
    hipDeviceSynchronize();

    for (int i = 0; i <= poly->degree; i++) {
        printf("%+.2lf", coefficients_host[i]);
        if (i < poly->degree) {
            printf("x^%d ", poly->degree - i);
        }
    }
    printf("\n");
    
    free(coefficients_host);
}

void partial_fraction_decomposition(Polynomial *numerator, Polynomial *denominator) {
    // Remain same as original function as this is not ideal for GPU computation
}

void free_polynomial(Polynomial *poly) {
    hipFree(poly->coefficients);
    free(poly);
}

int main() {
    srand(time(NULL));

    Polynomial *numerator, *denominator;

    numerator = generate_random_polynomial(200000, 200000);
    denominator = generate_random_polynomial(200000, 200000);

    printf("Generated numerator polynomial:\n");
    print_polynomial(numerator);

    printf("Generated denominator polynomial:\n");
    print_polynomial(denominator);

    clock_t start = clock();
    printf("The partial fraction decomposition is:\n");
    partial_fraction_decomposition(numerator, denominator);
    clock_t end = clock();
    double execution_time = (double)(end - start) / CLOCKS_PER_SEC;

    printf("\nExecution time: %lf seconds\n", execution_time);

    free_polynomial(numerator);
    free_polynomial(denominator);

    return 0;
}

